#include "hip/hip_runtime.h"
#define FP float
#define TW 32
#define TH 7

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <math.h>

__global__ void gpu_matmul_tiled(FP *a,FP *b, FP *c, int n, int p, int m) {

    FP cvalues[TH];
    __shared__ FP atile[TW][TW], btile[TH][TW][TW];

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int cols[TH]; // int col = tx + blockDim.x * blockIdx.x;
    int row = ty + blockDim.y * blockIdx.y;

    // init
    for (int i = 0; i < TH; ++i ) {
        cols[i] = tx + blockDim.x * (TH * blockIdx.x + i);
        cvalues[i] = 0;
    }

    // loop over tiles
    for (int i = 0; i < p / TW; ++ i) {
        atile[ty][tx] = a[row*p + i*TW + tx]; //Copy to shared memory
        for (int j = 0; j < TH; ++ j) {
            if (cols[j] < m) {  //Copy to shared memory
                btile[j][ty][tx] = b[(i * TW + ty) * m + cols[j]];
            }
        }
        __syncthreads();

        for (int j = 0; j < TH; ++ j) {
            for (int k = 0; k < TW; ++ k) {
                cvalues[j] += atile[ty][k] * btile[j][k][tx];
            }
        }
        __syncthreads();
    }

    // copy back to shared results
    for (int j = 0; j < TH; ++ j) {
        if (cols[j] < m) {
            c[row * m + cols[j]] = cvalues[j];
        }
    }
}

int main(int argc, char *argv[]) {

    int i, j; // loop counters

    int gpucount = 0; // Count of available GPUs
    int gpunum = 0; // Device number to use

    int n, p, m; // matrix dimension
    FP *a, *b, *c; // 1d-array representing matrices, where a(nxp), b(pxm), c(nxm).
    FP *dev_a, *dev_b, *dev_c;

    hipEvent_t start, stop; // using cuda events to measure time
    float elapsed_time_ms; // which is applicable for asynchronous code also
    hipError_t errorcode;

    // --------------------SET PARAMETERS AND DATA -----------------------

    errorcode = hipGetDeviceCount(&gpucount);
    if (errorcode == hipErrorNoDevice) {
        printf("No GPUs are visible\n");
        exit(-1);
    } else {
        printf("Device count = %d\n", gpucount);
    }

    if ((argc < 4) || (argc > 7)) {
        printf("Usage: matmul <matrix dim> <block dim> <grid dim> [<dev num>]\n");
        exit(-1);
    }

    n = atoi(argv[1]);
    p = atoi(argv[2]);
    m = atoi(argv[3]);

    if (argc == 5) {
        gpunum = atoi(argv[4]); // Device number
        if ((gpunum > 2) || (gpunum < 0)) {
            printf("Error, Device number must be 0, 1, or 2\n");
            exit(-1);
        }
    }
    hipSetDevice(gpunum);
    printf("Using device %d\n", gpunum);

    int Grid_dim_x, Grid_dim_y;
    int Block_dim_x, Block_dim_y;

    Block_dim_x = Block_dim_y = TW;
    Grid_dim_x = m / Block_dim_x / TH;
    Grid_dim_y = n / Block_dim_y;

    printf("Matrix Dimension = %d x %d\n", n, m);
    printf("Block_Dim = %d x %d, Grid_Dim = %d x %d\n", Block_dim_x, Block_dim_y, Grid_dim_x, Grid_dim_y);

    dim3 Grid(Grid_dim_x, Grid_dim_y); //Grid structure
    dim3 Block(Block_dim_x, Block_dim_y); //Block structure

    a = (FP *) malloc(n * p * sizeof(FP)); // dynamically allocated memory for arrays on host
    b = (FP *) malloc(p * m * sizeof(FP));
    c = (FP *) malloc(n * m * sizeof(FP)); // results from GPU

    srand(12345);
    for (i = 0; i < n; i++)
        for (j = 0; j < p; j++) {
            a[i * p + j] = (FP) rand() / (FP) RAND_MAX;
            // a[i * p + j] = (FP) i + j; // may be helpful for debugging
        }

    for (i = 0; i < p; i++)
        for (j = 0; j < m; j++) {
            b[i * m + j] = (FP) rand() / (FP) RAND_MAX;
            // b[i * m + j] = (FP) i + j; // may be helpful for debugging
        }

    // ------------- COMPUTATION DONE ON GPU ----------------------------

    hipMalloc((void **) &dev_a, n * p * sizeof(FP)); // allocate memory on device
    hipMalloc((void **) &dev_b, p * m * sizeof(FP));
    hipMalloc((void **) &dev_c, n * m * sizeof(FP));

    hipMemcpy(dev_a, a, n * p * sizeof(FP), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, p * m * sizeof(FP), hipMemcpyHostToDevice);

    hipEventCreate(&start); // instrument code to measure start time
    hipEventCreate(&stop);

    hipEventRecord(start, 0);

    // hipFuncSetCacheConfig(reinterpret_cast<const void*>(gpu_matmul_tiled), hipFuncCachePreferL1);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(gpu_matmul_tiled), hipFuncCachePreferShared);

    gpu_matmul_tiled << < Grid, Block >> > (dev_a, dev_b, dev_c, n, p, m);

    hipEventRecord(stop, 0); // instrument code to measure end time
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time_ms, start, stop);

    hipMemcpy(c, dev_c, n * m * sizeof(FP), hipMemcpyDeviceToHost);

    // Printing out diagonal to validate correctness
    for (i = 0; i < n; i+=32) {
        printf("%f ", c[i * m + i]);
        // }
    }
    printf("\n");

    printf("Time to calculate results on GPU: %f ms.\n", elapsed_time_ms); // exec. time

// -------------- clean up ---------------------------------------

    free(a);
    free(b);
    free(c);
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
