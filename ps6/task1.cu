#include "hip/hip_runtime.h"
#define FP float

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <math.h>

__global__ void gpu_matrixmult(FP *a,FP *b, FP *c, int n, int p, int m) {

    int row = threadIdx.y + blockDim.y * blockIdx.y;
    int col = threadIdx.x + blockDim.x * blockIdx.x;

    int indexb = col;
    int index = row * m + col;

    if(col < m && row < n) {
        c[index] = 0.;
        for (int indexa = row * p; indexa < (row * p + p); indexa ++, indexb += m)
            c[index] += a[indexa] * b[indexb];
    }

}

void cpu_matrixmult(FP *a,FP *b, FP *c, int n, int p, int m) {

    int i, j, k;
    for (k = 0; k < p; k++) {
        for (i = 0; i < n; i++) {
            FP r = a[i*p+k];
            for (j = 0; j < m; j++) {
                c[i*m+j] += r * b[k*m+j];
            }
        }
    }
}

int main(int argc, char *argv[]) {

    int i, j; // loop counters

    int gpucount = 0; // Count of available GPUs
    int gpunum = 0; // Device number to use

    int n, p, m; // matrix dimension
    FP *a, *b, *c; // 1d-array representing matrices, where a(nxp), b(pxm), c(nxm).
    FP *dev_a, *dev_b, *dev_c;

    hipEvent_t start, stop; // using cuda events to measure time
    float elapsed_time_ms; // which is applicable for asynchronous code also
    hipError_t errorcode;

    // --------------------SET PARAMETERS AND DATA -----------------------

    errorcode = hipGetDeviceCount(&gpucount);
    if (errorcode == hipErrorNoDevice) {
        printf("No GPUs are visible\n");
        exit(-1);
    } else {
        printf("Device count = %d\n", gpucount);
    }

    if ((argc < 4) || (argc > 7)) {
        printf("Usage: matmul <matrix dim> <block dim> <grid dim> [<dev num>]\n");
        exit(-1);
    }

    n = atoi(argv[1]);
    p = atoi(argv[2]);
    m = atoi(argv[3]);

//    Block_Dim = 32; // Block_dim ^ 2 <= 1024
//    Grid_Dim = 1; // Grid_dim * Block_dim >= MAX(m, n)
//    int max_size = (n > m) ? n : m;
//    while (Block_Dim * Grid_Dim < max_size) Grid_Dim *= 2;
//    Grid_Dim = 1024;

    if (argc == 5) {
        gpunum = atoi(argv[4]); // Device number
        if ((gpunum > 2) || (gpunum < 0)) {
            printf("Error, Device number must be 0, 1, or 2\n");
            exit(-1);
        }
    }
    hipSetDevice(gpunum);
    printf("Using device %d\n", gpunum);

    int Grid_dim_x, Grid_dim_y;
    int Block_dim_x, Block_dim_y;

    Block_dim_x = Block_dim_y = 32;
    Grid_dim_x = m / Block_dim_x;
    Grid_dim_y = n / Block_dim_y;

    printf("Matrix Dimension = %d x %d\n", n, m);
    printf("Block_Dim = %d x %d, Grid_Dim = %d x %d\n", Block_dim_x, Block_dim_y, Grid_dim_x, Grid_dim_y);

    dim3 Grid(Grid_dim_x, Grid_dim_y); //Grid structure
    dim3 Block(Block_dim_x, Block_dim_y); //Block structure

    a = (FP *) malloc(n * p * sizeof(FP)); // dynamically allocated memory for arrays on host
    b = (FP *) malloc(p * m * sizeof(FP));
    c = (FP *) malloc(n * m * sizeof(FP)); // results from GPU

    srand(12345);
    for (i = 0; i < n; i++)
        for (j = 0; j < p; j++) {
            a[i * p + j] = (FP) rand() / (FP) RAND_MAX;
            // a[i * p + j] = (FP) i + j; // may be helpful for debugging
        }

    for (i = 0; i < p; i++)
        for (j = 0; j < m; j++) {
            b[i * m + j] = (FP) rand() / (FP) RAND_MAX;
            // b[i * m + j] = (FP) i + j; // may be helpful for debugging
        }

    // ------------- COMPUTATION DONE ON GPU ----------------------------

    hipMalloc((void **) &dev_a, n * p * sizeof(FP)); // allocate memory on device
    hipMalloc((void **) &dev_b, p * m * sizeof(FP));
    hipMalloc((void **) &dev_c, n * m * sizeof(FP));

    hipMemcpy(dev_a, a, n * p * sizeof(FP), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, p * m * sizeof(FP), hipMemcpyHostToDevice);

    hipEventCreate(&start); // instrument code to measure start time
    hipEventCreate(&stop);

    hipEventRecord(start, 0);
    // hipEventSynchronize(start); // not needed

    gpu_matrixmult << < Grid, Block >> > (dev_a, dev_b, dev_c, n, p, m);

    hipEventRecord(stop, 0); // instrument code to measure end time
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time_ms, start, stop);

    hipMemcpy(c, dev_c, n * m * sizeof(FP), hipMemcpyDeviceToHost);
    for (i = 0; i < n; i+=32) {
        // for (j = 0; j < m; j++) {
        printf("%f ", c[i * m + i]);
        // }
    }
    printf("\n");

    printf("Time to calculate results on GPU: %f ms.\n", elapsed_time_ms); // exec. time

    // ------------- COMPUTATION DONE ON HOST CPU ----------------------------
    // DEBUGGING USE ONLY (AND FOR LIMITED NUMBERS OF TIMING RUNS)

    hipEventRecord(start, 0); // use same timing
    // hipEventSynchronize(start); // not needed

    for (i = 0; i < n * m; ++ i) c[i] = 0;
    cpu_matrixmult(a, b, c, n, p, m); // do calculation on host (NOTE: This computes the diff with GPU result.)

    hipEventRecord(stop, 0); // instrument code to measue end time
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time_ms, start, stop );

    printf("Time to calculate results on CPU: %f ms.\n", elapsed_time_ms); // exec. time

// ------------------- check device creates correct results -----------------

  double error, suma, sumb, sumc, ai, bi, ci;
  suma = 0.; sumb = 0; sumc = 0;
  for(i=0;i < n*p;i++) {
    ai = (double) a[i];
    suma += ai*ai;
  }
  for(i=0;i < p*m;i++) {
    bi = (double) b[i];
    sumb += bi*bi;
  }
  for(i=0;i < n*m;i++) {
    ci = (double) c[i];
    sumc += ci*ci;
  }
  suma = sqrt(suma);
  sumb = sqrt(sumb);
  sumc = sqrt(sumc);
  error =  sumc/(n*suma*sumb);
  printf("Scaled error between GPU and CPU: %e\n", error);

// -------------- clean up ---------------------------------------

    free(a);
    free(b);
    free(c);
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
